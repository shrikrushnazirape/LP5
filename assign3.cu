#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function for Min, Max, Sum, and Average operations
__global__ void reduction(float* input, int n, float* output_min, float* output_max, float* output_sum, float* output_avg) {
    __shared__ float shared_min;
    __shared__ float shared_max;
    __shared__ float shared_sum;
    
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize shared variables
    if (tid == 0) {
        shared_min = input[0];
        shared_max = input[0];
        shared_sum = 0;
    }
    __syncthreads();
    
    // Reduction loop
    while (i < n) {
        if (input[i] < shared_min) {
            shared_min = input[i];
        }
        if (input[i] > shared_max) {
            shared_max = input[i];
        }
        shared_sum += input[i];
        i += blockDim.x * gridDim.x;
    }
    
    // Reduce within block
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (shared_min > __shfl_down_sync(0xffffffff, shared_min, s)) {
                shared_min = __shfl_down_sync(0xffffffff, shared_min, s);
            }
            if (shared_max < __shfl_down_sync(0xffffffff, shared_max, s)) {
                shared_max = __shfl_down_sync(0xffffffff, shared_max, s);
            }
            shared_sum += __shfl_down_sync(0xffffffff, shared_sum, s);
        }
        __syncthreads();
    }
    
    // Write output variables
    if (tid == 0) {
        atomicMin(output_min, shared_min);
        atomicMax(output_max, shared_max);
        atomicAdd(output_sum, shared_sum);
        *output_avg = *output_sum / n;
    }
}

int main() {
    // Input array and its size
    float input[] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0};
    int n = sizeof(input) / sizeof(float);
    
    // Allocate memory on the device for the input array and the output variables
    float* d_input;
    hipMalloc(&d_input, n * sizeof(float));
    hipMemcpy(d_input, input, n * sizeof(float), hipMemcpyHostToDevice);
    
    float* d_output_min;
    hipMalloc(&d_output_min, sizeof(float));
    hipMemcpy(d_output_min, &input[0], sizeof(float), hipMemcpyHostToDevice);
    
    float* d_output_max;
    hipMalloc(&d_output_max, sizeof(float));
    hipMemcpy(d_output_max, &input[0], sizeof(float), hipMemcpyHostToDevice);
    
    float* d_output_sum;
    hipMalloc(&d_output_sum, sizeof(float));
    hipMemcpy(d_output_sum, &input[0], sizeof(float), hipMemcpyHostToDevice);
    
    float* d_output_avg;
    hipMalloc(&d_output_avg, sizeof(float));
    
    // Define block size and grid size
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    
    // Launch kernel function
// Copy output variables from device to host
// Pass output variables as arguments to the kernel function
reduction<<<grid_size, block_size>>>(d_input, n, d_output_min, d_output_max, d_output_sum, d_output_avg);

float output_min;
hipMemcpy(&output_min, d_output_min, sizeof(float), hipMemcpyDeviceToHost);

float output_max;
hipMemcpy(&output_max, d_output_max, sizeof(float), hipMemcpyDeviceToHost);

float output_sum;
hipMemcpy(&output_sum, d_output_sum, sizeof(float), hipMemcpyDeviceToHost);

float output_avg;
hipMemcpy(&output_avg, d_output_avg, sizeof(float), hipMemcpyDeviceToHost);

// Print output variables
printf("Min = %f\n", output_min);
printf("Max = %f\n", output_max);
printf("Sum = %f\n", output_sum);
printf("Average = %f\n", output_avg);

// Free memory on the device
hipFree(d_input);
hipFree(d_output_min);
hipFree(d_output_max);
hipFree(d_output_sum);
hipFree(d_output_avg);

return 0;
}


/*
nvcc -o reduction reduction.cu


./reduction



Input array: {1, 5, 2, 8, 4, 6, 3, 7}

Block size: 4

Grid size: 2


output : 
Min = 1.000000
Max = 8.000000
Sum = 36.000000
Average = 4.500000


*/